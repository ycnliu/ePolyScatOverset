#include "hip/hip_runtime.h"

// ApplyG0.cu
// Factored CUDA implementation of ApplyG0, ApplyG0UG, and ApplyG0EG in one file

#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <vector>
#include <iostream>
#include <cassert>
#include <cmath>

using Real = double;
using Complex = thrust::complex<double>;

// ---------------------------
// Utility Indexing Functions
// ---------------------------
#define IDX3D(ieg, ilh, irg, wmax, lp) (((ieg) * (wmax) * (lp)) + ((ilh) * (lp)) + (irg))

// ---------------------------
// Data Structures
// ---------------------------
namespace G0Overset {

struct GridParams {
    int N;        // Number of EG grids
    int wmax;     // Number of partial waves per grid
    int lp;       // Number of radial points per grid
};

struct GridUGParams {
    int N;        // Number of UG points
    int wmax;     // Number of partial waves per UG point
};

struct OperatorArraysEG {
    const Real* F;    // F_EG[N][wmax][lp]
    const Real* G;    // G_EG[N][wmax][lp]
    const Real* W;    // W_EG[N][wmax]
    const Real* Psi;  // Psi_EG[N][wmax][lp]
    const Real* grid_w; // grid weights per [N][lp]
    Real* Phi;        // Output: Phi_EG[N][wmax][lp]
};

struct OperatorArraysUG {
    const Real* F;
    const Real* G;
    const Real* W;
    const Real* Psi;
    const Real* grid_w;
    Real* Phi;
};

// ---------------------------
// CUDA Kernels
// ---------------------------

// EG kernel: full parallelization in (ieg, ilh, irg)
/**
 * @brief CUDA kernel to apply G0 operation on EG grid
 *
 * This kernel performs the G0 operation on an EG grid by
 * calculating contributions from F and G arrays and storing 
 * the result in the Phi array.
 *
 * @param N The number of grids.
 * @param wmax The maximum number of angular components per grid.
 * @param lp The number of radial points per grid.
 * @param F Pointer to the F array.
 * @param G Pointer to the G array.
 * @param W Pointer to the W array.
 * @param Psi Pointer to the Psi array.
 * @param grid_w Pointer to the grid weights array.
 * @param Phi Pointer to the output Phi array.
 */
__global__
void ApplyG0EG_kernel(int N, int wmax, int lp,
                      const Real* __restrict__ F,
                      const Real* __restrict__ G,
                      const Real* __restrict__ W,
                      const Real* __restrict__ Psi,
                      const Real* __restrict__ grid_w,
                      Real* __restrict__ Phi)
{
    // Calculate global thread indices for each dimension
    int ieg = blockIdx.z * blockDim.z + threadIdx.z;
    int ilh = blockIdx.y * blockDim.y + threadIdx.y;
    int irg = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure indices are within bounds
    if (ieg < N && ilh < wmax && irg < lp) {
        Real WF = 0.0; // Accumulator for contributions from F
        Real WG = 0.0; // Accumulator for contributions from G

        // Accumulate contributions over all radial points
        for (int jrg = 0; jrg < lp; ++jrg) {
            WF += F[IDX3D(ieg, ilh, jrg, wmax, lp)] * Psi[IDX3D(ieg, ilh, jrg, wmax, lp)];
            WG += G[IDX3D(ieg, ilh, jrg, wmax, lp)] * Psi[IDX3D(ieg, ilh, jrg, wmax, lp)];
        }

        // Calculate the wronskian factor for the current radial point
        Real wf = sqrt(grid_w[ieg * lp + irg]);

        // Retrieve the weight value from W
        Real wval = W[ieg * wmax + ilh];

        // Compute the result for the current grid point and store in Phi
        Phi[IDX3D(ieg, ilh, irg, wmax, lp)] = wf * wval * (WF * G[IDX3D(ieg, ilh, irg, wmax, lp)] + WG * F[IDX3D(ieg, ilh, irg, wmax, lp)]);
    }
}

// UG kernel: full parallelization in (irg, ilh)
/**
 * @brief CUDA kernel to apply G0 operation on UG grid
 *
 * This kernel performs the G0 operation on an UG grid by
 * calculating the sum of contributions from F and G arrays
 * and storing the result in the Phi array.
 *
 * @param N The number of radial grid points.
 * @param wmax The maximum number of angular components.
 * @param F Pointer to the F array.
 * @param G Pointer to the G array.
 * @param W Pointer to the W array.
 * @param Psi Pointer to the Psi array.
 * @param grid_w Pointer to the grid_w array.
 * @param Phi Pointer to the output Phi array.
 */
__global__
void ApplyG0UG_kernel(int N, int wmax,
                      const Real* __restrict__ F,
                      const Real* __restrict__ G,
                      const Real* __restrict__ W,
                      const Real* __restrict__ Psi,
                      const Real* __restrict__ grid_w,
                      Real* __restrict__ Phi)
{
    int irg = blockIdx.x * blockDim.x + threadIdx.x;
    int ilh = blockIdx.y * blockDim.y + threadIdx.y;

    if (irg < N && ilh < wmax) {
        Real sum = 0;

        // Accumulate contributions from F before current radial point
        for (int jrg = 0; jrg <= irg; ++jrg)
            sum += F[jrg * wmax + ilh] * Psi[jrg * wmax + ilh];

        // Accumulate contributions from G after current radial point
        for (int jrg = irg; jrg < N; ++jrg)
            sum += G[jrg * wmax + ilh] * Psi[jrg * wmax + ilh];

        // Apply weights and store the result in Phi
        Phi[irg * wmax + ilh] = sqrt(grid_w[irg]) * W[ilh] * sum;
    }
}

// ---------------------------
// Host-side Wrappers
// ---------------------------

// EG grid host-side wrapper
void ApplyG0EG_GPU(const GridParams& params, const OperatorArraysEG& arrays)
{
    int x = std::min(256, params.lp);
    int y = std::min(16, params.wmax);
    dim3 block(x, y);
    dim3 grid((params.lp + block.x - 1) / block.x,
              (params.wmax + block.y - 1) / block.y,
              params.N);

    ApplyG0EG_kernel<<<grid, block, 0, 0>>>(
        params.N, params.wmax, params.lp,
        arrays.F, arrays.G, arrays.W, arrays.Psi, arrays.grid_w, arrays.Phi
    );
    hipDeviceSynchronize();
}

// UG grid host-side wrapper
void ApplyG0UG_GPU(const GridUGParams& params, const OperatorArraysUG& arrays)
{
    int x = std::min(1024, params.N);
    int y = std::min(16, params.wmax);
    dim3 block(x, y);
    dim3 grid((params.N + block.x - 1) / block.x,
              (params.wmax + block.y - 1) / block.y);

    ApplyG0UG_kernel<<<grid, block>>>(
        params.N, params.wmax,
        arrays.F, arrays.G, arrays.W, arrays.Psi, arrays.grid_w, arrays.Phi
    );
    hipDeviceSynchronize();
}

// Main dispatcher: ApplyG0 (applies either or both, then could combine results)
void ApplyG0(const GridParams& eg_params, const OperatorArraysEG& eg_arrays,
             const GridUGParams& ug_params, const OperatorArraysUG& ug_arrays,
             bool use_eg, bool use_ug)
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    if (use_eg)
        ApplyG0EG_GPU(eg_params, eg_arrays, stream);
    if (use_ug)
        ApplyG0UG_GPU(ug_params, ug_arrays, stream);

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
}

// EG grid host-side wrapper
void ApplyG0EG_GPU(const GridParams& params, const OperatorArraysEG& arrays, hipStream_t stream)
{
    dim3 block(16, 8);
    dim3 grid((params.lp + block.x - 1) / block.x,
              (params.wmax + block.y - 1) / block.y,
              params.N);

    ApplyG0EG_kernel<<<grid, block, 0, stream>>>(
        params.N, params.wmax, params.lp,
        arrays.F, arrays.G, arrays.W, arrays.Psi, arrays.grid_w, arrays.Phi
    );
}

// UG grid host-side wrapper
void ApplyG0UG_GPU(const GridUGParams& params, const OperatorArraysUG& arrays, hipStream_t stream)
{
    dim3 block(32, 8);
    dim3 grid((params.N + block.x - 1) / block.x,
              (params.wmax + block.y - 1) / block.y);

    ApplyG0UG_kernel<<<grid, block, 0, stream>>>(
        params.N, params.wmax,
        arrays.F, arrays.G, arrays.W, arrays.Psi, arrays.grid_w, arrays.Phi
    );
}

} // namespace G0Overset

// ---------------------------
// Example Main/Test Harness
// ---------------------------
void test_comm()
{
    const int N = 2;        // EG grids
    const int wmax = 4;     // Partial waves
    const int lp = 8;       // Radial points
    const int N_UG = 8;     // Ubergrid points

    std::vector<Real> F_EG(N * wmax * lp), G_EG(N * wmax * lp), W_EG(N * wmax), Psi_EG(N * wmax * lp), grid_w(N * lp), Phi_EG(N * wmax * lp);
    std::vector<Real> F_UG(N_UG * wmax), G_UG(N_UG * wmax), W_UG(wmax), Psi_UG(N_UG * wmax), grid_w_UG(N_UG), Phi_UG(N_UG * wmax);

    Real *d_F_EG, *d_G_EG, *d_W_EG, *d_Psi_EG, *d_grid_w, *d_Phi_EG;
    hipMalloc(&d_F_EG, F_EG.size() * sizeof(Real));
    hipMalloc(&d_G_EG, G_EG.size() * sizeof(Real));
    hipMalloc(&d_W_EG, W_EG.size() * sizeof(Real));
    hipMalloc(&d_Psi_EG, Psi_EG.size() * sizeof(Real));
    hipMalloc(&d_grid_w, grid_w.size() * sizeof(Real));
    hipMalloc(&d_Phi_EG, Phi_EG.size() * sizeof(Real));

    Real *d_F_UG, *d_G_UG, *d_W_UG, *d_Psi_UG, *d_grid_w_UG, *d_Phi_UG;
    hipMalloc(&d_F_UG, F_UG.size() * sizeof(Real));
    hipMalloc(&d_G_UG, G_UG.size() * sizeof(Real));
    hipMalloc(&d_W_UG, W_UG.size() * sizeof(Real));
    hipMalloc(&d_Psi_UG, Psi_UG.size() * sizeof(Real));
    hipMalloc(&d_grid_w_UG, grid_w_UG.size() * sizeof(Real));
    hipMalloc(&d_Phi_UG, Phi_UG.size() * sizeof(Real));

    hipMemcpy(d_F_EG, F_EG.data(), F_EG.size() * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_G_EG, G_EG.data(), G_EG.size() * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_W_EG, W_EG.data(), W_EG.size() * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_Psi_EG, Psi_EG.data(), Psi_EG.size() * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_grid_w, grid_w.data(), grid_w.size() * sizeof(Real), hipMemcpyHostToDevice);

    hipMemcpy(d_F_UG, F_UG.data(), F_UG.size() * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_G_UG, G_UG.data(), G_UG.size() * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_W_UG, W_UG.data(), W_UG.size() * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_Psi_UG, Psi_UG.data(), Psi_UG.size() * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_grid_w_UG, grid_w_UG.data(), grid_w_UG.size() * sizeof(Real), hipMemcpyHostToDevice);

    GridParams eg_params{N, wmax, lp};
    GridUGParams ug_params{N_UG, wmax};
    OperatorArraysEG eg_arrays{d_F_EG, d_G_EG, d_W_EG, d_Psi_EG, d_grid_w, d_Phi_EG};
    OperatorArraysUG ug_arrays{d_F_UG, d_G_UG, d_W_UG, d_Psi_UG, d_grid_w_UG, d_Phi_UG};

    ApplyG0(eg_params, eg_arrays, ug_params, ug_arrays, /*use_eg=*/true, /*use_ug=*/true);

    hipMemcpy(Phi_EG.data(), d_Phi_EG, Phi_EG.size() * sizeof(Real), hipMemcpyDeviceToHost);
    hipMemcpy(Phi_UG.data(), d_Phi_UG, Phi_UG.size() * sizeof(Real), hipMemcpyDeviceToHost);

    std::cout << "Phi_EG result: ";
    for (int i = 0; i < std::min(10, int(Phi_EG.size())); ++i)
        std::cout << Phi_EG[i] << " ";
    std::cout << std::endl;

    std::cout << "Phi_UG result: ";
    for (int i = 0; i < std::min(10, int(Phi_UG.size())); ++i)
        std::cout << Phi_UG[i] << " ";
    std::cout << std::endl;

    hipFree(d_F_EG); hipFree(d_G_EG); hipFree(d_W_EG); hipFree(d_Psi_EG); hipFree(d_grid_w); hipFree(d_Phi_EG);
    hipFree(d_F_UG); hipFree(d_G_UG); hipFree(d_W_UG); hipFree(d_Psi_UG); hipFree(d_grid_w_UG); hipFree(d_Phi_UG);
}
